
#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>
#define HEAVY 10000

hipError_t CounterCuda(int * ArrNumbers, int n, int myid, int * results);
void FreeMethod(int * arr, int * result);


__device__ double f_gpu(int i) {
	int j;
	double value;
	double result = 0;

	for (j = 1; j < HEAVY; j++) {
		value = (i + 1)*(j % 10);
		result += cos(value);
	}
	return cos(result);
}

__global__ void CounterKernel(int *Arr_dev,  int * results_dev)
{
	int i = blockIdx.x * 1024 +  threadIdx.x;
	
		if (f_gpu(Arr_dev[i]) > 0)
		{
			results_dev[i] = 1;
		}
		else
		{
			results_dev[i] = 0;
		}
	

	
}



// Helper function for using CUDA to add vectors in parallel.

hipError_t CounterCuda(int * ArrNumbers, int n, int myid, int * results)
{
	int *dev_ArrNumbers = 0;
	int *dev_results = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		FreeMethod(dev_ArrNumbers, dev_results);
		return cudaStatus;

	}


	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_ArrNumbers, (n/2) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		FreeMethod(dev_ArrNumbers, dev_results);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_results, (n/2) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		FreeMethod(dev_ArrNumbers, dev_results);
		return cudaStatus;
	}


	// process 1 contains the first half of array

	// process 0 second half array 

	if (myid == 0)
	{											// second half
		cudaStatus = hipMemcpy(dev_ArrNumbers, ArrNumbers + (n/2), (n / 2) * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			FreeMethod(dev_ArrNumbers, dev_results);
			return cudaStatus;
		}
	}
	else
	{
		if (myid == 1)
		{											// first half
			cudaStatus = hipMemcpy(dev_ArrNumbers, ArrNumbers, (n / 2) * sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				FreeMethod(dev_ArrNumbers, dev_results);
				return cudaStatus;
			}
		}
	}

	int dim_blocks = ((n/2) / 1024) + 1; // 49

	CounterKernel <<< dim_blocks , 1024 >>>(dev_ArrNumbers,dev_results);

	// Copy input vectors from host memory to GPU buffers.

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CounterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		FreeMethod(dev_ArrNumbers, dev_results);
		return cudaStatus;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		FreeMethod(dev_ArrNumbers, dev_results);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, dev_results, (n/2) * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMethod(dev_ArrNumbers, dev_results);
		return cudaStatus;
	}
	

	FreeMethod(dev_ArrNumbers, dev_results);

	return cudaStatus;



}


void FreeMethod(int * arr, int * result)
{
	hipFree(arr);
	hipFree(result);
}
